#include "hip/hip_runtime.h"
#include "clarke_wright.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>
#include <cmath>
#include <iostream>
#include <vector>
#include <cstdint> // Inclure cstdint pour les types de taille fixe

using namespace std;
using namespace thrust;

// Fonction pour calculer la distance euclidienne entre deux points
__device__ float euclidean_distance(float x1, float y1, float x2, float y2) {
    return sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1));
}

// Noyau pour calculer les distances entre tous les points
__global__ void compute_distances(const float* locations, float* distances, int64_t n) {
    int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < n) {
        float x1 = locations[i * 2];
        float y1 = locations[i * 2 + 1];
        float x2 = locations[j * 2];
        float y2 = locations[j * 2 + 1];
        distances[i * n + j] = euclidean_distance(x1, y1, x2, y2);
    }
}

// Fonction Clarke-Wright pour calculer les routes
vector<vector<int64_t>> clarke_wright(const thrust::device_vector<float>& d_locations) {
    int64_t n = d_locations.size() / 2 - 1;
    thrust::device_vector<float> d_distances(n * n);

    // Définir les dimensions du noyau
    dim3 blockDim(16, 16);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);

    // Lancer le noyau pour remplir les distances
    compute_distances<<<gridDim, blockDim>>>(thrust::raw_pointer_cast(d_locations.data()), thrust::raw_pointer_cast(d_distances.data()), n);
    hipDeviceSynchronize();

    thrust::host_vector<float> h_distances = d_distances;
    vector<thrust::tuple<float, int64_t, int64_t>> savings;

    // Calculer les économies et trier
    for (int64_t i = 1; i <= n; ++i) {
        for (int64_t j = i + 1; j <= n; ++j) {
            float saving = h_distances[i * (n + 1) + 0] + h_distances[j * (n + 1) + 0] - h_distances[i * (n + 1) + j];
            savings.push_back(thrust::make_tuple(saving, i, j));
        }
    }

    // Trier les économies en ordre décroissant
    thrust::sort(savings.begin(), savings.end(), thrust::greater<thrust::tuple<float, int64_t, int64_t>>());

    vector<vector<int64_t>> routes(n);
    for (int64_t i = 0; i < n; ++i) {
        routes[i].push_back(i + 1);
    }

    // Construire les routes
    for (const auto& saving : savings) {
        float s;
        int64_t i, j;
        thrust::tie(s, i, j) = saving;
        if (!routes[i - 1].empty() && !routes[j - 1].empty() && routes[i - 1] != routes[j - 1]) {
            routes[i - 1].insert(routes[i - 1].end(), routes[j - 1].begin(), routes[j - 1].end());
            routes[j - 1].clear();
        }
    }

    vector<vector<int64_t>> final_routes;
    for (const auto& route : routes) {
        if (!route.empty()) {
            final_routes.push_back(route);
        }
    }

    return final_routes;
}
