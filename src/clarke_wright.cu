#include "hip/hip_runtime.h"
#include "clarke_wright.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>
#include <cmath>
#include <iostream>
#include <vector>

using namespace std;
using namespace thrust;

// Fonction pour calculer la distance euclidienne entre deux points
__device__ float euclidean_distance(float x1, float y1, float x2, float y2) {
    return sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1));
}

// Noyau pour calculer les distances entre tous les points
__global__ void compute_distances(const float* locations, float* distances, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < n) {  // Corrige la condition pour éviter les dépassements
        float x1 = locations[i * 2];
        float y1 = locations[i * 2 + 1];
        float x2 = locations[j * 2];
        float y2 = locations[j * 2 + 1];
        distances[i * n + j] = euclidean_distance(x1, y1, x2, y2);  // Correction de l'indexation
    }
}

// Fonction Clarke-Wright pour calculer les routes
vector<vector<int>> clarke_wright(const thrust::device_vector<float>& d_locations) {
    int n = d_locations.size() / 2 - 1;
    thrust::device_vector<float> d_distances(n * n);  // Correction de la taille du vecteur

    // Définir les dimensions du noyau
    dim3 blockDim(16, 16);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (n + blockDim.y - 1) / blockDim.y);

    // Lancer le noyau pour remplir les distances
    compute_distances<<<gridDim, blockDim>>>(thrust::raw_pointer_cast(d_locations.data()), thrust::raw_pointer_cast(d_distances.data()), n);
    hipDeviceSynchronize();

    thrust::host_vector<float> h_distances = d_distances;
    vector<thrust::tuple<float, int, int>> savings;

    // Calculer les économies et trier
    for (int i = 1; i <= n; ++i) {
        for (int j = i + 1; j <= n; ++j) {
           float saving = h_distances[i * (n + 1) + 0] + h_distances[j * (n + 1) + 0] - h_distances[i * (n + 1) + j];
            savings.push_back(thrust::make_tuple(saving, i, j));
        }
    }

    // Trier les économies en ordre décroissant
    thrust::sort(savings.begin(), savings.end(), thrust::greater<thrust::tuple<float, int, int>>());

    vector<vector<int>> routes(n);
    for (int i = 0; i < n; ++i) {
        routes[i].push_back(i + 1);
    }

    // Construire les routes
    for (const auto& saving : savings) {
        float s;
        int i, j;
        thrust::tie(s, i, j) = saving;
        if (!routes[i - 1].empty() && !routes[j - 1].empty() && routes[i - 1] != routes[j - 1]) {
            routes[i - 1].insert(routes[i - 1].end(), routes[j - 1].begin(), routes[j - 1].end());
            routes[j - 1].clear();
        }
    }

    vector<vector<int>> final_routes;
    for (const auto& route : routes) {
        if (!route.empty()) {
            final_routes.push_back(route);
        }
    }

    return final_routes;
}
